#include "hip/hip_runtime.h"
#include "kernel.h"

// #include <cuda/std/span>  // Should work out of the box


namespace kernel {

int cuda_malloc_managed(void** data, std::size_t size) { return hipMallocManaged(data, size); }

int cuda_free(void* ptr) { return hipFree(ptr); }

int cuda_malloc(void** d_data, std::size_t size) { return hipMalloc(d_data, size); }

int cuda_memcpy(void* to, void* from, std::size_t size, cuda_memcpy_kind kind) {
    hipError_t err;
    switch (kind) {
        case cuda_memcpy_kind::hipMemcpyHostToDevice:
        err = hipMemcpy(to, from, size, hipMemcpyHostToDevice);
        break;
        case cuda_memcpy_kind::hipMemcpyDeviceToHost:
        err = hipMemcpy(to, from, size, hipMemcpyDeviceToHost);
        break;
        default:
        err = hipError_t(-1);
    }
    return err;
}

__global__ void add(int* data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) data[idx] += 1;
}

float apply(int* data, int N) {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    add<<<1, 1>>>(data, N);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    return milliseconds;
}

}  // namespace kernel
